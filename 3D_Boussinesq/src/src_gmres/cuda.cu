#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "../kernels.h"
#include "../functions.h"

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////\
////////////////////////

extern "C" void timestep_cuda_(double2 *xii,
			       double2 *eta,
			       double2 *zet,
			       double2 *rho,
			       double *kx,
			       double *ky,
			       double *kz,
			       double *Time, 
			       double *TSTART,
			       double *Tstep,
			       double *AMPFOR,
			       double *DELT,
			       double *ResThresh,
			       double *KFY,
			       double *V2,
			       double *Ri,
			       double *Sc,
			       double *Theta,
			       double *alpha,
			       double *Dtarget,
			       int *ikF,
			       int *ikN,
			       int *LC,
			       int * IKTX,
			       int * IKTY,
			       int * IKTZ,
			       int * KTZ,
			       int *NKT,
			       int * NX,
			       int * NY,
			       int *NZ,
			       int *NOUT,
			       int *NOUTV,
			       int *statsFLAG,
			       int *RecFLAG,
			       int *adaptFLAG,
			       int *RANK
			       )
{
  
  // Define CPU variables
  printf("\n In timestep_cuda \n");
  ny = *NY;
  nx = *NX;
  nz = *NZ;
  kfy= *KFY;
  nkt = *NKT;
  nr  = (*NY)*(*NX)*(*NZ);
  nx2 = (*NX)/2 +1;
  n2 = nx2*(*NY)*(*NZ);
  in = 1.0/nr;
  dtarget = *Dtarget;
  ri = *Ri;
  delt = *DELT;
  nOut = *NOUT;
  tme = *TSTART;
  tstart = *TSTART;
  ampfor = -*AMPFOR;
  v2 = *V2;
  ResidualThreshold = *ResThresh;
  rank = *RANK;
  printf("Tstep %e \n",*Tstep);
  printf("ampfor %e \n",ampfor);
  printf("delt %e \n",delt);
  printf("v2 %e \n",v2);
  printf("Ri %e\n",*Ri);
  printf("Sc %e\n",*Sc);
  printf("theta %e \n",*Theta);
  printf("kfy %e \n",kfy);

  double IN = 1.0/nr;
  sinTh = sin(*Theta);
  cosTh = cos(*Theta);
  size_t avail,total;
  
  // Define global device variables
  int *d_ikF, *d_ikN, *d_LL;
  hipfftDoubleComplex *d_xii,*d_eta,*d_zet,*d_rho,*d_UK, *d_VK, *d_WK,*d_rk;
  hipfftDoubleComplex *d_x0,*d_x1,*d_x2,*d_x3,*d_e0,*d_e1,*d_e2,*d_e3,*d_z0,*d_z1,*d_z2,*d_z3,*d_r0,*d_r1,*d_r2,*d_r3;
  hipfftDoubleReal *d_xir,*d_etr,*d_ztr,*d_rx,*d_ry,*d_rz,*d_UR,*d_VR,*d_WR,*d_ro;
  hipfftHandle PlanZ2D,PlanD2Z;

  // Allocate global memory on GPU. (Constant memory does not need allocating) 	
  (hipMalloc((void**)&d_xii,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_x0,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_x1,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_x2,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_x3,sizeof(hipfftDoubleComplex)*(nkt)));

  (hipMalloc((void**)&d_eta,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_e0,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_e1,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_e2,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_e3,sizeof(hipfftDoubleComplex)*(nkt)));

  (hipMalloc((void**)&d_zet,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_z0,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_z1,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_z2,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_z3,sizeof(hipfftDoubleComplex)*(nkt)));


  (hipMalloc((void**)&d_rho,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_r0,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_r1,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_r2,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_r3,sizeof(hipfftDoubleComplex)*(nkt)));

  (hipMalloc((void**)&d_UK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_VK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_WK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_rk,sizeof(hipfftDoubleComplex)*(nkt)));
  
  (hipMalloc((void**)&d_xir,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_etr,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_ztr,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_rx,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_ry,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_rz,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_ro,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_UR,sizeof(hipfftDoubleReal)*nr));
  (hipMalloc((void**)&d_VR,sizeof(hipfftDoubleReal)*nr));
  (hipMalloc((void**)&d_WR,sizeof(hipfftDoubleReal)*nr));
  
  (hipMalloc((void**)&d_kx,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_ky,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_kz,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_LL,sizeof(int)*nkt));
  (hipMalloc((void**)&d_ikN,sizeof(int)*nkt));
  (hipMalloc((void**)&d_ikF,sizeof(int)*n2));
  
  // Copy state data to GPU global memory 
  (hipMemcpy(d_xii,xii,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_eta,eta,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_zet,zet,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_rho,rho,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  //Set up various arrays to enable generic kernel calls
  //i.e. calculate indexing for padding either side of FFTs, wavenumber arrays, mask, and timestep arrays.
  // This must be done on CPU for scalability (large problems violate max threads per block)
  (hipMemcpy(d_kx,kx,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_ky,ky,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_kz,kz,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_ikF,ikF, sizeof(int)*n2,hipMemcpyHostToDevice));
  (hipMemcpy(d_ikN,ikN,sizeof(int)*nkt,hipMemcpyHostToDevice));
  (hipMemcpy(d_LL,LC,sizeof(int)*nkt,hipMemcpyHostToDevice));

  // Copy constsant parameters to GPU constant memory
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IN),&IN,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_AMPFOR),AMPFOR,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_DELT),&delt,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_v2),V2,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_Ri),Ri,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_Sc),Sc,sizeof(double))); 
  (hipMemcpyToSymbol(HIP_SYMBOL(d_SinTh),&sinTh,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_CosTh),&cosTh,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_KFY),KFY,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IKTX),IKTX,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IKTY),IKTY,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IKTZ),IKTZ,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_KTZ),KTZ,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_NX),NX,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_NX2),&nx2,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_NY),NY,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_NZ),NY,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_OR),&nr,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_OK),&nkt,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_O2),&n2,sizeof(int)));
    
  //Set FFT Plans
  (hipfftPlan3d(&PlanZ2D,*NZ,*NY,*NX,HIPFFT_Z2D));
  (hipfftPlan3d(&PlanD2Z,*NZ,*NY,*NX,HIPFFT_D2Z));
  // Do a check of global memory use
  avail =0;
  total = 0;
  hipMemGetInfo(&avail,&total);
  
  // Do an initial set of velocity coeffs. Subsequently this occurs at the end of stepping kernels
  setVelocity<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_kx,d_ky,d_kz,d_LL);
 
  // **************************
  //STEPPING STARTS HERE
  // **************************
  fflush(stdout);
  int timestep=0;
  while(*Tstep-tme > delt){
    timestep++;

    KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);

    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W	 
    RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
    Step1<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_x1,d_e1,d_z1,d_r1,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);
    
    KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);
    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W
    RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
    Step1<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_x2,d_e2,d_z2,d_r2,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);	 

    KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);
    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W
    RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
    Step2<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_x3,d_e3,d_z3,d_r3,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);	 

    KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);
    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W
    RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
    Step<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_x1,d_e1,d_z1,d_r1,d_x2,d_e2,d_z2,d_r2,d_x3,d_e3,d_z3,d_r3,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);	 

    *Time = *Time + delt;    // Increase time
    tme += delt;
    
  }
  if( tme != *Tstep){
    delt = *Tstep-tme;
    printf("adjusting final timestep dt = %e \n",delt);
    (hipMemcpyToSymbol(HIP_SYMBOL(d_DELT),&delt,sizeof(double)));
    KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);

    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W	 
    RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
    Step1<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_x1,d_e1,d_z1,d_r1,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);
    
    KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);
    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W
    RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
    Step1<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_x2,d_e2,d_z2,d_r2,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);	 

    KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);
    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W
    RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
    Step2<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_x3,d_e3,d_z3,d_r3,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);	 

    KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);
    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W
    RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
    Step<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_x1,d_e1,d_z1,d_r1,d_x2,d_e2,d_z2,d_r2,d_x3,d_e3,d_z3,d_r3,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);	 

  }
  fflush(stdout);

    // Copy state off GPU
  (hipMemcpy(xii, d_xii, sizeof(hipfftDoubleComplex)*nkt, hipMemcpyDeviceToHost));
  (hipMemcpy(eta, d_eta, sizeof(hipfftDoubleComplex)*nkt, hipMemcpyDeviceToHost));
  (hipMemcpy(zet, d_zet, sizeof(hipfftDoubleComplex)*nkt, hipMemcpyDeviceToHost));
  (hipMemcpy(rho, d_rho, sizeof(hipfftDoubleComplex)*nkt, hipMemcpyDeviceToHost));

  printf("pdfs and averages done \n"); 
  // Free GPU global memory
  (hipFree(d_xii));
  (hipFree(d_eta));
  (hipFree(d_zet));
  (hipFree(d_rho));
  (hipFree(d_UK));
  (hipFree(d_VK));
  (hipFree(d_WK));
  (hipFree(d_rk));
  
  (hipFree(d_x0));	
  (hipFree(d_e0));	
  (hipFree(d_z0));	
  (hipFree(d_r0));	

  (hipFree(d_x1));	
  (hipFree(d_e1));	
  (hipFree(d_z1));	
  (hipFree(d_r1));	

  (hipFree(d_x2));	
  (hipFree(d_e2));	
  (hipFree(d_z2));	
  (hipFree(d_r2));	

  (hipFree(d_x3));	
  (hipFree(d_e3));	
  (hipFree(d_z3));	
  (hipFree(d_r3));	

  (hipFree(d_xir));	
  (hipFree(d_etr));	
  (hipFree(d_ztr));	
  (hipFree(d_rx));	
  (hipFree(d_ry));	
  (hipFree(d_rz));	
  (hipFree(d_ro));	
  (hipFree(d_UR));
  (hipFree(d_VR));
  (hipFree(d_WR));
  
  (hipFree(d_LL));
  (hipFree(d_ikF));
  (hipFree(d_ikN));
  (hipFree(d_kx));
  (hipFree(d_ky));
  (hipFree(d_kz));
  printf("GPU free done \n"); 
  //Destroy fft plans
  (hipfftDestroy(PlanZ2D));
  (hipfftDestroy(PlanD2Z));
   
  printf("time stepping done \n"); 
  fflush(stdout);

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////\
////////////////////////

extern "C" void timederiv_cuda_(double2 *xii,
			       double2 *eta,
			       double2 *zet,
			       double2 *rho,
			       double *kx,
			       double *ky,
			       double *kz,
			       double *Time, 
			       double *TSTART,
			       double *Tstep,
			       double *AMPFOR,
			       double *DELT,
			       double *ResThresh,
			       double *KFY,
			       double *V2,
			       double *Ri,
			       double *Sc,
			       double *Theta,
			       double *alpha,
			       double *Dtarget,
			       int *ikF,
			       int *ikN,
			       int *LC,
			       int * IKTX,
			       int * IKTY,
			       int * IKTZ,
			       int * KTZ,
			       int *NKT,
			       int * NX,
			       int * NY,
			       int *NZ,
			       int *NOUT,
			       int *NOUTV,
			       int *statsFLAG,
			       int *RecFLAG,
			       int *adaptFLAG,
			       int *RANK
			       )
{
  
  // Define CPU variables
  printf("\n In timestep_cuda \n");
  ny = *NY;
  nx = *NX;
  nz = *NZ;
  kfy= *KFY;
  nkt = *NKT;
  nr  = (*NY)*(*NX)*(*NZ);
  nx2 = (*NX)/2 +1;
  n2 = nx2*(*NY)*(*NZ);
  in = 1.0/nr;
  dtarget = *Dtarget;
  ri = *Ri;
  double nStop = max(*Tstep/(*DELT),1.0);
  if(nStop**DELT != *Tstep){
    delt = *Tstep/nStop;
      }else{
    delt = *DELT;  
  }
  nOut = *NOUT;
  tme = *TSTART;
  tstart = *TSTART;
  ampfor = -*AMPFOR;
  v2 = *V2;
  ResidualThreshold = *ResThresh;
  rank = *RANK;
  printf("Tstep %e \n",*Tstep);
  printf("ampfor %e \n",ampfor);
  printf("delt %e \n",delt);
  printf("v2 %e \n",v2);
  printf("Ri %e\n",*Ri);
  printf("Sc %e\n",*Sc);
  printf("theta %e \n",*Theta);
  printf("kfy %e \n",kfy);

  double IN = 1.0/nr;
  sinTh = sin(*Theta);
  cosTh = cos(*Theta);
  printf("sin(theta) %e \n",sinTh);
  printf("cos(theta) %e \n",cosTh);
  
  // Define global device variables
  int *d_ikF, *d_ikN, *d_LL;
  hipfftDoubleComplex *d_xii,*d_eta,*d_zet,*d_rho,*d_UK, *d_VK, *d_WK,*d_rk;
  hipfftDoubleComplex *d_x0,*d_e0,*d_z0,*d_r0;
  hipfftDoubleReal *d_xir,*d_etr,*d_ztr,*d_rx,*d_ry,*d_rz,*d_UR,*d_VR,*d_WR,*d_ro;
  hipfftHandle PlanZ2D,PlanD2Z;

  // Allocate global memory on GPU. (Constant memory does not need allocating) 	
  (hipMalloc((void**)&d_xii,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_x0,sizeof(hipfftDoubleComplex)*(nkt)));

  (hipMalloc((void**)&d_eta,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_e0,sizeof(hipfftDoubleComplex)*(nkt)));

  (hipMalloc((void**)&d_zet,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_z0,sizeof(hipfftDoubleComplex)*(nkt)));

  (hipMalloc((void**)&d_rho,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_r0,sizeof(hipfftDoubleComplex)*(nkt)));

  (hipMalloc((void**)&d_UK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_VK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_WK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_rk,sizeof(hipfftDoubleComplex)*(nkt)));
  
  (hipMalloc((void**)&d_xir,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_etr,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_ztr,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_rx,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_ry,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_rz,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_ro,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_UR,sizeof(hipfftDoubleReal)*nr));
  (hipMalloc((void**)&d_VR,sizeof(hipfftDoubleReal)*nr));
  (hipMalloc((void**)&d_WR,sizeof(hipfftDoubleReal)*nr));
  
  (hipMalloc((void**)&d_kx,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_ky,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_kz,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_LL,sizeof(int)*nkt));
  (hipMalloc((void**)&d_ikN,sizeof(int)*nkt));
  (hipMalloc((void**)&d_ikF,sizeof(int)*n2));
  
  // Copy state data to GPU global memory 
  (hipMemcpy(d_xii,xii,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_eta,eta,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_zet,zet,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_rho,rho,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  //Set up various arrays to enable generic kernel calls
  //i.e. calculate indexing for padding either side of FFTs, wavenumber arrays, mask, and timestep arrays.
  // This must be done on CPU for scalability (large problems violate max threads per block)
  (hipMemcpy(d_kx,kx,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_ky,ky,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_kz,kz,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_ikF,ikF, sizeof(int)*n2,hipMemcpyHostToDevice));
  (hipMemcpy(d_ikN,ikN,sizeof(int)*nkt,hipMemcpyHostToDevice));
  (hipMemcpy(d_LL,LC,sizeof(int)*nkt,hipMemcpyHostToDevice));

  // Copy constsant parameters to GPU constant memory
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IN),&IN,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_AMPFOR),AMPFOR,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_DELT),&delt,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_v2),V2,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_Ri),Ri,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_Sc),Sc,sizeof(double))); 
  (hipMemcpyToSymbol(HIP_SYMBOL(d_SinTh),&sinTh,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_CosTh),&cosTh,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_KFY),KFY,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IKTX),IKTX,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IKTY),IKTY,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IKTZ),IKTZ,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_KTZ),KTZ,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_NX),NX,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_NX2),&nx2,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_NY),NY,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_NZ),NY,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_OR),&nr,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_OK),&nkt,sizeof(int)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_O2),&n2,sizeof(int)));
    
  //Set FFT Plans
  (hipfftPlan3d(&PlanZ2D,*NZ,*NY,*NX,HIPFFT_Z2D));
  (hipfftPlan3d(&PlanD2Z,*NZ,*NY,*NX,HIPFFT_D2Z));
  // Do an initial set of velocity coeffs. Subsequently this occurs at the end of stepping kernels
  setVelocity<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_kx,d_ky,d_kz,d_LL);
  KR_FFT_ALL(d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz,PlanZ2D,d_ikF,n2);
  
  multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,d_rx,d_ry,d_rz); // real space convolution, result in U,V,W	 
  RK_FFT(d_UK,d_VK,d_WK,d_rk,d_UR,d_VR,d_WR,d_rx,PlanD2Z,d_ikN,d_LL,n2);
  RHS<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_rho,d_x0,d_e0,d_z0,d_r0,d_UK,d_VK,d_WK,d_rk,d_kx,d_ky,d_kz,d_LL);
  hipDeviceSynchronize();
    // Copy state off GPU
  (hipMemcpy(xii, d_x0, sizeof(hipfftDoubleComplex)*nkt, hipMemcpyDeviceToHost));
  (hipMemcpy(eta, d_e0, sizeof(hipfftDoubleComplex)*nkt, hipMemcpyDeviceToHost));
  (hipMemcpy(zet, d_z0, sizeof(hipfftDoubleComplex)*nkt, hipMemcpyDeviceToHost));
  (hipMemcpy(rho, d_r0, sizeof(hipfftDoubleComplex)*nkt, hipMemcpyDeviceToHost));

  printf("pdfs and averages done \n"); 
  // Free GPU global memory
  (hipFree(d_xii));
  (hipFree(d_eta));
  (hipFree(d_zet));
  (hipFree(d_rho));
  (hipFree(d_UK));
  (hipFree(d_VK));
  (hipFree(d_WK));
  (hipFree(d_rk));
  
  (hipFree(d_x0));	
  (hipFree(d_e0));	
  (hipFree(d_z0));	
  (hipFree(d_r0));	

  (hipFree(d_xir));	
  (hipFree(d_etr));	
  (hipFree(d_ztr));	
  (hipFree(d_rx));	
  (hipFree(d_ry));	
  (hipFree(d_rz));	
  (hipFree(d_ro));	
  (hipFree(d_UR));
  (hipFree(d_VR));
  (hipFree(d_WR));
  
  (hipFree(d_LL));
  (hipFree(d_ikF));
  (hipFree(d_ikN));
  (hipFree(d_kx));
  (hipFree(d_ky));
  (hipFree(d_kz));
  //Destroy fft plans
  (hipfftDestroy(PlanZ2D));
  (hipfftDestroy(PlanD2Z));
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
