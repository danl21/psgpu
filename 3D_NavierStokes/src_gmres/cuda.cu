#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include "kernels.h"
#include "functions.h"

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////\
////////////////////////

extern "C" void timestep_cuda_(double2* xii,
			       double2* eta,
			       double2* zet,
			       double* nuzn,
			       double* nu1,
			       double* kx,
			       double* ky,
			       double* kz,
			       double* Time, 
			       double* TSTART,
			       double* AMPFOR,
			       double* DELT,
			       int* KFY,
			       double* V2,
			       int* ikF,
			       int* ikN,
			       int* LC,
			       int* NSTOP
			       )
{
  
  // Define CPU variables
  n2 = nx2*ny*nz;
  in = 1.0/nr;
  tme = 0.0;
  tstart = *TSTART;
  delt = *DELT;  
  ampfor = *AMPFOR;
  v2 = *V2;
  kfy = double(*KFY);

  // Define global device variables
  int *d_ikF, *d_ikN, *d_LL;
  hipfftDoubleComplex *d_xii,*d_eta,*d_zet, *d_UK, *d_VK, *d_WK, *d_RHX,*d_RHY,*d_RHZ;
  hipfftDoubleReal *d_xir,*d_etr,*d_ztr, *d_UR,*d_VR,*d_WR;
  double *d_nuzn,*d_nu1;
  hipfftHandle PlanZ2D,PlanD2Z;
  
  // Allocate global memory on GPU. (Constant memory does not need allocating) 	
  (hipMalloc((void**)&d_xii,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_eta,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_zet,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_UK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_VK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_WK,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_RHX,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_RHY,sizeof(hipfftDoubleComplex)*(nkt)));
  (hipMalloc((void**)&d_RHZ,sizeof(hipfftDoubleComplex)*(nkt)));
  
  (hipMalloc((void**)&d_xir,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_etr,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_ztr,sizeof(hipfftDoubleReal)*(nr)));
  (hipMalloc((void**)&d_UR,sizeof(hipfftDoubleReal)*nr));
  (hipMalloc((void**)&d_VR,sizeof(hipfftDoubleReal)*nr));
  (hipMalloc((void**)&d_WR,sizeof(hipfftDoubleReal)*nr));
  
  (hipMalloc((void**)&d_nuzn,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_nu1,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_kx,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_ky,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_kz,sizeof(double)*(nkt)));
  (hipMalloc((void**)&d_LL,sizeof(int)*nkt));
  (hipMalloc((void**)&d_ikN,sizeof(int)*nkt));
  (hipMalloc((void**)&d_ikF,sizeof(int)*n2));
  
  // Copy state data to GPU global memory 
  (hipMemcpy(d_xii,xii,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_eta,eta,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_zet,zet,sizeof(hipfftDoubleComplex)*(nkt),hipMemcpyHostToDevice));
  //Set up various arrays to enable generic kernel calls
  //i.e. calculate indexing for padding either side of FFTs, wavenumber arrays, mask, and timestep arrays.
  // This must be done on CPU for scalability (large problems violate max threads per block)

  (hipMemcpy(d_nuzn,nuzn,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_nu1,nu1,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_kx,kx,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_ky,ky,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_kz,kz,sizeof(double)*(nkt),hipMemcpyHostToDevice));
  (hipMemcpy(d_ikF,ikF, sizeof(int)*n2,hipMemcpyHostToDevice));
  (hipMemcpy(d_ikN,ikN,sizeof(int)*nkt,hipMemcpyHostToDevice));
  (hipMemcpy(d_LL,LC,sizeof(int)*nkt,hipMemcpyHostToDevice));

  // Copy constsant parameters to GPU constant memory
  (hipMemcpyToSymbol(HIP_SYMBOL(d_IN),&in,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_AMPFOR),AMPFOR,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_DELT),DELT,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_v2),V2,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_KFY),&kfy,sizeof(double)));
  (hipMemcpyToSymbol(HIP_SYMBOL(d_O2),&n2,sizeof(int)));
    
  //Set FFT Plans
  (hipfftPlan3d(&PlanZ2D,nz,ny,nx,HIPFFT_Z2D));
  (hipfftPlan3d(&PlanD2Z,nz,ny,nx,HIPFFT_D2Z));

  setVelocity<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_UK,d_VK,d_WK,d_kx,d_ky,d_kz,d_LL);
 
  // **************************
  //STEPPING STARTS HERE
  // **************************
  fflush(stdout);
  
  for(int NT=0; NT<*NSTOP; NT++){
    
    KR_FFT_ALL(d_xii,d_eta,d_zet,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,PlanZ2D,d_ikF,n2);

    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR); // real space convolution, result in U,V,W
	 
    RK_FFT(d_UK,d_VK,d_WK,d_UR,d_VR,d_WR,PlanD2Z,d_ikN,n2);
	  
    // Predictor step: prestep now does the end of 'convol', the step and resets velocity coeffs
    preStep<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_UK,d_VK,d_WK,d_RHX,d_RHY,d_RHZ,d_nuzn,d_nu1,d_kx,d_ky,d_kz,d_LL);
    
    KR_FFT_ALL(d_xii,d_eta,d_zet,d_UK,d_VK,d_WK,d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR,PlanZ2D,d_ikF,n2);

    multReal<<<nblocks,nthreads>>>(d_xir,d_etr,d_ztr,d_UR,d_VR,d_WR); // real space convolution, result in U,V,W
	 
    RK_FFT(d_UK,d_VK,d_WK,d_UR,d_VR,d_WR,PlanD2Z,d_ikN,n2);

    *Time = *TSTART + (NT+1)*(*DELT);    // Increase time
    tme += *DELT;

    //Correction step: corStep is analagous in structure to prestep.
    corStep<<<nblocks,nthreads>>>(d_xii,d_eta,d_zet,d_UK,d_VK,d_WK,d_RHX,d_RHY,d_RHZ,d_nuzn,d_nu1,d_kx,d_ky,d_kz,d_LL);

  }
  fflush(stdout);

    // Copy state off GPU

  (hipMemcpy(xii, d_xii, sizeof(hipfftDoubleComplex)*(nkt), hipMemcpyDeviceToHost));
  (hipMemcpy(eta, d_eta, sizeof(hipfftDoubleComplex)*(nkt), hipMemcpyDeviceToHost));
  (hipMemcpy(zet, d_zet, sizeof(hipfftDoubleComplex)*(nkt), hipMemcpyDeviceToHost));

  // Free GPU global memory
  (hipFree(d_xii));
  (hipFree(d_eta));
  (hipFree(d_zet));
  (hipFree(d_UK));
  (hipFree(d_VK));
  (hipFree(d_WK));
  (hipFree(d_RHX));
  (hipFree(d_RHY));
  (hipFree(d_RHZ));
  
  (hipFree(d_xir));	
  (hipFree(d_etr));	
  (hipFree(d_ztr));	
  (hipFree(d_UR));
  (hipFree(d_VR));
  (hipFree(d_WR));
  
  (hipFree(d_nuzn));
  (hipFree(d_nu1));
  (hipFree(d_LL));
  (hipFree(d_ikF));
  (hipFree(d_ikN));
  (hipFree(d_kx));
  (hipFree(d_ky));
  (hipFree(d_kz));

  //Destroy fft plans
  (hipfftDestroy(PlanZ2D));
  (hipfftDestroy(PlanD2Z));

  fflush(stdout);

}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
